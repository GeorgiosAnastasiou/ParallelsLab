#include "hip/hip_runtime.h"
__device__ int get_tid(){
	return blockIdx.x * blockDim.x + threadIdx.x; /* TODO: copy me from naive version... */
}

/* square of Euclid distance between two multi-dimensional points using column-base format */
	__host__ __device__ inline static
double euclid_dist_2_transpose(int numCoords,
		int    numObjs,
		int    numClusters,
		double *objects,     // [numCoords][numObjs]
		double *clusters,    // [numCoords][numClusters]
		int    objectId,
		int    clusterId)
{
	int i;
	double ans=0.0;

	/* TODO: Copy me from transpose version*/

	/* TODO: Calculate the euclid_dist of elem=objectId of objects from elem=clusterId from clusters, but for column-base format!!! */
	for (i = 0; i < numCoords; i++)
		ans += (objects[i*numObjs + objectId] - clusters[i*numClusters + clusterId]) *
			(objects[i*numObjs + objectId] - clusters[i*numClusters + clusterId]);

	return(ans);
}
	__global__ static
void find_nearest_cluster(int numCoords,
		int numObjs,
		int numClusters,
		double *objects,           //  [numCoords][numObjs]
		double *deviceClusters,    //  [numCoords][numClusters]
		int *deviceMembership,          //  [numObjs]
		double *devdelta)
{
	extern __shared__ double shmemClusters[];

	/* TODO: Copy deviceClusters to shmemClusters so they can be accessed faster. 
BEWARE: Make sure operations is complete before any thread continues... */
	for (int i = 0; i < numClusters; i++) {
		for (int j = 0; j < numCoords; j++) {
			shmemClusters[j * numClusters + i] = deviceClusters[j * numClusters + i];
		}
	}

	__syncthreads();

	/* Get the global ID of the thread. */
	int tid = get_tid(); 

	/* TODO: Maybe something is missing here... should all threads run this? */
	if (tid < numObjs) { // was 1
		int   index, i;
		double dist, min_dist;

		/* find the cluster id that has min distance to object */
		index = 0;
		/* TODO: call min_dist = euclid_dist_2(...) with correct objectId/clusterId */
		min_dist = euclid_dist_2_transpose(numCoords, numObjs, numClusters, objects, shmemClusters, tid, 0); // was empty

		for (i=1; i<numClusters; i++) {
			/* TODO: call dist = euclid_dist_2(...) with correct objectId/clusterId */
			dist = euclid_dist_2_transpose(numCoords, numObjs, numClusters, objects, shmemClusters, tid, i); // was empty

			/* no need square root */
			if (dist < min_dist) { /* find the min and its array index */
				min_dist = dist;
				index    = i;
			}
		}

		if (deviceMembership[tid] != index) {
			/* TODO: Maybe something is missing here... is this write safe? */
			atomicAdd(devdelta, 1.0); // was (*devdelta)+= 1.0;
		}

		/* assign the deviceMembership to object objectId */
		deviceMembership[tid] = index;
	}
}
...
/*	Define the shared memory needed per block.
	- BEWARE: We can overrun our shared memory here if there are too many
	clusters or too many coordinates! 
	- This can lead to occupancy problems or even inability to run. 
	- Your exercise implementation is not requested to account for that (e.g. always assume deviceClusters fit in shmemClusters */
const unsigned int clusterBlockSharedDataSize = numClusters*numCoords*sizeof(double);
...
do {
	timing_internal = wtime(); 
	/* GPU part: calculate new memberships */
#ifdef TIMER_ANALYSIS
	time_start = wtime();
#endif
	/* TODO: Copy clusters to deviceClusters
	   checkCuda(hipMemcpy(...)); */
	checkCuda(hipMemcpy(deviceClusters, dimClusters[0],
				clusterBlockSharedDataSize, hipMemcpyHostToDevice));

	checkCuda(hipMemset(dev_delta_ptr, 0, sizeof(double)));          
#ifdef TIMER_ANALYSIS
	TIME(cpu_gpu_time);
#endif
	//printf("Launching find_nearest_cluster Kernel with grid_size = %d, block_size = %d, shared_mem = %d KB\n", numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize/1000);
	find_nearest_cluster
		<<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
		(numCoords, numObjs, numClusters,
		 deviceObjects, deviceClusters, deviceMembership, dev_delta_ptr);

	hipDeviceSynchronize(); checkLastCudaError();
	//printf("Kernels complete for itter %d, updating data in CPU\n", loop);
#ifdef TIMER_ANALYSIS
	TIME(gpu_time);
#endif
	/* TODO: Copy deviceMembership to membership
	   checkCuda(hipMemcpy(...)); */
	checkCuda(hipMemcpy(membership, deviceMembership,
				numObjs*sizeof(int), hipMemcpyDeviceToHost));

	/* TODO: Copy dev_delta_ptr to &delta
	   checkCuda(hipMemcpy(...)); */
	checkCuda(hipMemcpy(&delta, dev_delta_ptr,
				sizeof(double), hipMemcpyDeviceToHost));
#ifdef TIMER_ANALYSIS
	TIME(gpu_cpu_time);
#endif
	/* CPU part: Update cluster centers*/
	for (i=0; i<numObjs; i++) {
		/* find the array index of nestest cluster center */
		index = membership[i];

		/* update new cluster centers : sum of objects located within */
		newClusterSize[index]++;
		for (j=0; j<numCoords; j++)
			newClusters[j][index] += objects[i*numCoords + j];
	}
	/* average the sum and replace old cluster centers with newClusters */
	for (i=0; i<numClusters; i++) {
		for (j=0; j<numCoords; j++) {
			if (newClusterSize[i] > 0)
				dimClusters[j][i] = newClusters[j][i] / newClusterSize[i];
			newClusters[j][i] = 0.0;   /* set back to 0 */
		}
		newClusterSize[i] = 0;   /* set back to 0 */
	}
	delta /= numObjs;
	loop++;
	...
} while (delta > threshold && loop < loop_threshold);

/*TODO: Update clusters using dimClusters. Be carefull of layout!!! clusters[numClusters][numCoords] vs dimClusters[numCoords][numClusters] */
for (i = 0; i < numCoords; i++) {
	for (j = 0; j < numClusters; j++) {
		clusters[j*numCoords + i] = dimClusters[i][j];
	}
}

