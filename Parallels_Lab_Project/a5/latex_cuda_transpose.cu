#include "hip/hip_runtime.h"
__device__ int get_tid(){
	return blockIdx.x * blockDim.x + threadIdx.x; /* TODO: copy me from naive version... */
}

/* square of Euclid distance between two multi-dimensional points using column-base format */
	__host__ __device__ inline static
double euclid_dist_2_transpose(int numCoords,
		int    numObjs,
		int    numClusters,
		double *objects,     // [numCoords][numObjs]
		double *clusters,    // [numCoords][numClusters]
		int    objectId,
		int    clusterId)
{
	int i;
	double ans=0.0;

	/* TODO: Calculate the euclid_dist of elem=objectId of objects from elem=clusterId from clusters, but for column-base format!!! */
	for (i = 0; i < numCoords; i++)
		ans += (objects[i*numObjs + objectId] - clusters[i*numClusters + clusterId]) *
			(objects[i*numObjs + objectId] - clusters[i*numClusters + clusterId]);

	return(ans);
}

	__global__ static
void find_nearest_cluster(int numCoords,
		int numObjs,
		int numClusters,
		double *objects,           //  [numCoords][numObjs]
		double *deviceClusters,    //  [numCoords][numClusters]
		int *membership,          //  [numObjs]
		double *devdelta)
{
	/* TODO: copy me from naive version... */

	/* Get the global ID of the thread. */
	int tid = get_tid(); 

	/* TODO: Maybe something is missing here... should all threads run this? */
	if (tid < numObjs) { // was 1
		int   index, i;
		double dist, min_dist;

		/* find the cluster id that has min distance to object */
		index = 0;
		/* TODO: call min_dist = euclid_dist_2(...) with correct objectId/clusterId */
		min_dist = euclid_dist_2_transpose(numCoords, numObjs, numClusters, objects, deviceClusters, tid, 0); // was empty

		for (i=1; i<numClusters; i++) {
			/* TODO: call dist = euclid_dist_2(...) with correct objectId/clusterId */
			dist = euclid_dist_2_transpose(numCoords, numObjs, numClusters, objects, deviceClusters, tid, i); // was empty

			/* no need square root */
			if (dist < min_dist) { /* find the min and its array index */
				min_dist = dist;
				index    = i;
			}
		}

		if (membership[tid] != index) {
			/* TODO: Maybe something is missing here... is this write safe? */
			atomicAdd(devdelta, 1.0); // was (*devdelta)+= 1.0;
		}

		/* assign the deviceMembership to object objectId */
		membership[tid] = index;
	}
}
...
/* TODO: Transpose dims */
double  **dimObjects = NULL; //calloc_2d(...) -> [numCoords][numObjs]
double  **dimClusters = NULL;  //calloc_2d(...) -> [numCoords][numClusters]
double  **newClusters = NULL;  //calloc_2d(...) -> [numCoords][numClusters]
dimObjects  = (double**) calloc_2d(numCoords, numObjs, sizeof(double));
dimClusters = (double**) calloc_2d(numCoords, numClusters, sizeof(double));
newClusters = (double**) calloc_2d(numCoords, numClusters, sizeof(double));
...

//  TODO: Copy objects given in [numObjs][numCoords] layout to new
//  [numCoords][numObjs] layout
for (i = 0; i < numCoords; i++) {
	for (j = 0; j < numObjs; j++) {
		dimObjects[i][j] = objects[j*numCoords + i];
	}
}
...
do {
	timing_internal = wtime();

	/* GPU part: calculate new memberships */

#ifdef TIMER_ANALYSIS
	time_start = wtime();
#endif

	/* TODO: Copy clusters to deviceClusters
	   checkCuda(hipMemcpy(...)); */
	checkCuda(hipMemcpy(deviceClusters, dimClusters[0],
				numClusters*numCoords*sizeof(double), hipMemcpyHostToDevice));

	checkCuda(hipMemset(dev_delta_ptr, 0, sizeof(double)));          

#ifdef TIMER_ANALYSIS
	TIME(cpu_gpu_time);
#endif

	find_nearest_cluster
		<<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
		(numCoords, numObjs, numClusters,
		 deviceObjects, deviceClusters, deviceMembership, dev_delta_ptr);

	hipDeviceSynchronize(); checkLastCudaError();

#ifdef TIMER_ANALYSIS
	TIME(gpu_time);
#endif

	/* TODO: Copy deviceMembership to membership
	   checkCuda(hipMemcpy(...)); */
	checkCuda(hipMemcpy(membership, deviceMembership,
				numObjs*sizeof(int), hipMemcpyDeviceToHost));

	/* TODO: Copy dev_delta_ptr to &delta
	   checkCuda(hipMemcpy(...)); */
	checkCuda(hipMemcpy(&delta, dev_delta_ptr,
				sizeof(double), hipMemcpyDeviceToHost));

#ifdef TIMER_ANALYSIS
	TIME(gpu_cpu_time);
#endif

	/* CPU part: Update cluster centers*/

	for (i=0; i<numObjs; i++) {
		/* find the array index of nestest cluster center */
		index = membership[i];

		/* update new cluster centers : sum of objects located within */
		newClusterSize[index]++;
		for (j=0; j<numCoords; j++)
			newClusters[j][index] += objects[i*numCoords + j];
	}

	/* average the sum and replace old cluster centers with newClusters */
	for (i=0; i<numClusters; i++) {
		for (j=0; j<numCoords; j++) {
			if (newClusterSize[i] > 0)
				dimClusters[j][i] = newClusters[j][i] / newClusterSize[i];
			newClusters[j][i] = 0.0;   /* set back to 0 */
		}
		newClusterSize[i] = 0;   /* set back to 0 */
	}

	delta /= numObjs;
	//printf("delta is %f - ", delta);
	loop++; 
	//printf("completed loop %d\n", loop);
	...
} while (delta > threshold && loop < loop_threshold);

/*TODO: Update clusters using dimClusters. Be carefull of layout!!! clusters[numClusters][numCoords] vs dimClusters[numCoords][numClusters] */
for (i = 0; i < numCoords; i++) {
	for (j = 0; j < numClusters; j++) {
		clusters[j*numCoords + i] = dimClusters[i][j];
	}
}

