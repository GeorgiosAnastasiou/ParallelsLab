#include "hip/hip_runtime.h"
__global__ static
void find_nearest_cluster(int numCoords,
        int numObjs,
        int numClusters,
        double *deviceobjects,           //  [numCoords][numObjs]
        /*                          
                                    TODO: If you choose to do (some of) the new centroid calculation here, you will need some extra parameters here (from "update_centroids").
         */                          
        int *devicenewClusterSize,           //  [numClusters]
        double *devicenewClusters,    //  [numCoords][numClusters]
                                      //added above two
        double *deviceClusters,    //  [numCoords][numClusters]
        int *deviceMembership,          //  [numObjs]
        double *devdelta)
{
    extern __shared__ double shmemClusters[];

    /* TODO: copy me from shared version... */
    . . . 
    /* TODO: additional steps for calculating new centroids in GPU? */
    atomicAdd(&devicenewClusterSize[index], 1);
    for (i = 0; i < numCoords; ++i)
        atomicAdd(&devicenewClusters[i*numClusters + index], deviceobjects[i*numObjs + tid]);
}

    __global__ static
void update_centroids(int numCoords,
        int numClusters,
        int *devicenewClusterSize,           //  [numClusters]
        double *devicenewClusters,    //  [numCoords][numClusters]
        double *deviceClusters)    //  [numCoords][numClusters])
{

    /* TODO: additional steps for calculating new centroids in GPU? */
    //was empty
    const int tid = get_tid();
    if (tid >= numClusters*numCoords) return;
    int cluster = tid % numClusters; // tid = coord*numClusters + cluster, which makes access bellow fast af
    if (devicenewClusterSize[cluster] > 0)
        deviceClusters[tid] = devicenewClusters[tid]/devicenewClusterSize[cluster];
    devicenewClusters[tid] = 0.0;
    // apparently synchronizing here doesn't change the results (also each thread does it lol, could add if (coord == 0))
    devicenewClusterSize[cluster] = 0;
}
