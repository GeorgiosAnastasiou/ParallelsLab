#include "hip/hip_runtime.h"
__device__ int get_tid(){
	return blockIdx.x * blockDim.x + threadIdx.x; /* TODO: copy me from naive version... */
}

/* square of Euclid distance between two multi-dimensional points */
__host__ __device__ inline static
double euclid_dist_2(int    numCoords,
                    int    numObjs,
                    int    numClusters,
                    double *objects,     // [numObjs][numCoords]
                    double *clusters,    // [numClusters][numCoords]
                    int    objectId,
                    int    clusterId)
{
    int i;
    double ans=0.0;

	/* TODO: Calculate the euclid_dist of elem=objectId of objects from elem=clusterId from clusters*/
    // was empty
    for (i=0; i<numCoords; i++)
        ans += (objects[objectId*numCoords + i] - clusters[clusterId*numCoords + i]) *
               (objects[objectId*numCoords + i] - clusters[clusterId*numCoords + i]);

    return(ans);
}

__global__ static
void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          double *objects,           //  [numObjs][numCoords]
                          double *deviceClusters,    //  [numClusters][numCoords]
                          int *deviceMembership,          //  [numObjs]
                          double *devdelta)
{

	/* Get the global ID of the thread. */
    int tid = get_tid(); 

	/* TODO: Maybe something is missing here... should all threads run this? */
    if (tid < numObjs) { // was 1
        int   index, i;
        double dist, min_dist;

        /* find the cluster id that has min distance to object */
        index = 0;
        /* TODO: call min_dist = euclid_dist_2(...) with correct objectId/clusterId */
        min_dist = euclid_dist_2(numCoords, numObjs, numClusters, objects, deviceClusters, tid, 0); // was empty

        for (i=1; i<numClusters; i++) {
            /* TODO: call dist = euclid_dist_2(...) with correct objectId/clusterId */
            dist = euclid_dist_2(numCoords, numObjs, numClusters, objects, deviceClusters, tid, i); // was empty
 
            /* no need square root */
            if (dist < min_dist) { /* find the min and its array index */
                min_dist = dist;
                index    = i;
            }
        }

        if (deviceMembership[tid] != index) {
        	/* TODO: Maybe something is missing here... is this write safe? */
            atomicAdd(devdelta, 1.0); // was (*devdelta)+= 1.0;
        }

        /* assign the deviceMembership to object objectId */
        deviceMembership[tid] = index;
    }
}


    const unsigned int numThreadsPerClusterBlock = (numObjs > blockSize)? blockSize: numObjs;
    /* TODO: Calculate Grid size, e.g. number of blocks. */
    const unsigned int numClusterBlocks = (numObjs + numThreadsPerClusterBlock - 1) / numThreadsPerClusterBlock; // was -1
    const unsigned int clusterBlockSharedDataSize = 0;


    do {
        timing_internal = wtime(); 
 
		/* GPU part: calculate new memberships */
        #ifdef TIMER_ANALYSIS
        time_start = wtime();
        #endif
		        
        /* TODO: Copy clusters to deviceClusters
        checkCuda(hipMemcpy(...)); */
        checkCuda(hipMemcpy(deviceClusters, clusters,
              numClusters*numCoords*sizeof(double), hipMemcpyHostToDevice));
        
        checkCuda(hipMemset(dev_delta_ptr, 0, sizeof(double)));

        #ifdef TIMER_ANALYSIS
        TIME(cpu_gpu_time);
        #endif

        find_nearest_cluster
            <<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
            (numCoords, numObjs, numClusters,
             deviceObjects, deviceClusters, deviceMembership, dev_delta_ptr);

        hipDeviceSynchronize(); checkLastCudaError();

        #ifdef TIMER_ANALYSIS
        TIME(gpu_time);
        #endif
		
		/* TODO: Copy deviceMembership to membership
        checkCuda(hipMemcpy(...)); */
        checkCuda(hipMemcpy(membership, deviceMembership,
              numObjs*sizeof(int), hipMemcpyDeviceToHost));
    
    	/* TODO: Copy dev_delta_ptr to &delta
        checkCuda(hipMemcpy(...)); */
        checkCuda(hipMemcpy(&delta, dev_delta_ptr,
              sizeof(double), hipMemcpyDeviceToHost));

        #ifdef TIMER_ANALYSIS
        TIME(gpu_cpu_time);
        #endif

		/* CPU part: Update cluster centers*/
		  		
        for (i=0; i<numObjs; i++) {
            /* find the array index of nestest cluster center */
            index = membership[i];
			
            /* update new cluster centers : sum of objects located within */
            newClusterSize[index]++;
            for (j=0; j<numCoords; j++)
                newClusters[index][j] += objects[i*numCoords + j];
        }
        
        /* average the sum and replace old cluster centers with newClusters */
        for (i=0; i<numClusters; i++) {
            for (j=0; j<numCoords; j++) {
                if (newClusterSize[i] > 0)
                    clusters[i*numCoords + j] = newClusters[i][j] / newClusterSize[i];
                newClusters[i][j] = 0.0;   /* set back to 0 */
            }
            newClusterSize[i] = 0;   /* set back to 0 */
        }

        delta /= numObjs;
       	//printf("delta is %f - ", delta);
        loop++; 
        ...     
    } while (delta > threshold && loop < loop_threshold);
    
